#include "hip/hip_runtime.h"
#include "cuda_multMatVec.cuh"


__global__ void MultMatVec_GPU_GlobalMemoryWithoutRegister_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		vecY[ tid * ELEM_PER_VECTOR ] = 0.0f; // No use register
		for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
		{
			vecY[ tid * ELEM_PER_VECTOR + i ] += matA[ i ][ j ] * vecX[ tid * ELEM_PER_VECTOR + j ];
		}
	}
}

__global__ void MultMatVec_GPU_GlobalMemoryWithRegister_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	float result;
	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		result = 0.0f;
		for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
		{
			result += matA[ i ][ j ] * vecX[ tid * ELEM_PER_VECTOR + j ];
		}
		vecY[ tid * ELEM_PER_VECTOR + i ] = result;
	}
}

__global__ void MultMatVec_GPU_GlobalMemoryWithoutRegister_Element32ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

	vecY[ tid ] = 0.0f;
	for( unsigned j = 0; j < blockDim.x; ++j )
	{
		vecY[ tid ] += matA[ threadIdx.x ][ j ] * vecX[ blockIdx.x * ELEM_PER_VECTOR + j ];
	}
}

__global__ void MultMatVec_GPU_GlobalMemoryWithRegister_Element32ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	float result = 0.0f;
	for( unsigned j = 0; j < blockDim.x; ++j )
	{
		result += matA[ threadIdx.x ][ j ] * vecX[ blockIdx.x * ELEM_PER_VECTOR + j ];
	}
	vecY[ blockIdx.x * blockDim.x + threadIdx.x ] = result;
}

__global__ void MultMatVec_GPU_GlobalMemoryWithRegister_Element1024ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += matA[ eid ][ j ] * vecX[ vid * ELEM_PER_VECTOR + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}

__global__ void MultMatVec_GPU_GlobalMemoryWithAtomic_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		vecY[ vid * ELEM_PER_VECTOR + i ] = 0.0f;
		atomicAdd( &vecY[ vid * ELEM_PER_VECTOR + i ], matA[ i ][ eid ] * vecX[ vid * ELEM_PER_VECTOR + eid ] );
	}
}

__global__ void MultMatVec_GPU_StridedGlobalMemory_Element1024ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned n = gridDim.x * blockDim.x;
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned stride = ( n / ELEM_PER_VECTOR );

	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid / stride;

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += matA[ eid ][ j ] * vecX[ vid * ELEM_PER_VECTOR + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}

__constant__ float constantMatA[ ELEM_PER_VECTOR ][ ELEM_PER_VECTOR ];
void GenerateConstantMatrix( IN float( *matA )[ ELEM_PER_VECTOR ] )
{
	hipMemcpyToSymbol(HIP_SYMBOL( constantMatA), matA, sizeof( float ) * ELEM_PER_VECTOR * ELEM_PER_VECTOR );
}

__global__ void MultMatVec_GPU_SimpleConstantMatrixWithRegister_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	float result;
	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		result = 0.0f;
		for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
		{
			result += constantMatA[ i ][ j ] * vecX[ tid * ELEM_PER_VECTOR + j ];
		}
		vecY[ tid * ELEM_PER_VECTOR + i ] = result;
	}
}

__global__ void MultMatVec_GPU_SimpleConstantMatrixWithAtomic_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		vecY[ vid * ELEM_PER_VECTOR + i ] = 0.0f;
		atomicAdd( &vecY[ vid * ELEM_PER_VECTOR + i ], constantMatA[ i ][ eid ] * vecX[ vid * ELEM_PER_VECTOR + eid ] );
	}
}

__global__ void MultMatVec_GPU_SimpleConstantMatrixWithRegister_Element1024ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += constantMatA[ eid ][ j ] * vecX[ vid * ELEM_PER_VECTOR + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}

__global__ void MultMatVec_GPU_StridedConstantMatrixWithRegister_Element1024ThreadsPerBlock( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	// BLOCK DIM: 128 (CC:3.5)
	unsigned n = gridDim.x * blockDim.x;
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned stride = ( n / ELEM_PER_VECTOR );
	
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid / stride;

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += constantMatA[ eid ][ j ] * vecX[ vid * ELEM_PER_VECTOR + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}

// �̰� ���� �����غ� �ڵ尡 ������ �ȵ�
__global__ void MultMatVec_GPU_BroadcastConstantMatrixWithRegister_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	float result;
	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		result = 0.0f;
		for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
		{
			result += constantMatA[ i ][ j ] * vecX[ tid * ELEM_PER_VECTOR + j ];
		}
		vecY[ tid * ELEM_PER_VECTOR + i ] = result;
	}
}

__global__ void MultMatVec_GPU_GlobalMemoryWithRegister_Vector_Direction( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

	float result = 0.0f;
	
	
}

//extern __shared__ unsigned char sharedBuffer[];
//__shared__ float sharedVecX[];
__global__ void MultMatVec_GPU_SharedMemoryWithAtomic_Vector( OUT float* vecY, IN float( *matA )[ ELEM_PER_VECTOR ], IN float* vecX )
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	unsigned ratio = 1024 / blockDim.x; // num of elements in 32x32 Matrix
	__shared__ float sharedMatA[ 1024 ];
	__shared__ float sharedVecY[ 128 ];

	// normally access
	// GTX 690, 29.x ms
	//for( unsigned i = 0; i < ratio; ++i )
	//	sharedMatA[ threadIdx.x * ratio + i ] = ( ( float* )matA )[ threadIdx.x * ratio + i ];
	//__syncthreads();

	// coalesed access
	// GTX 690, 25.x ms
	int accessID = ( threadIdx.x / ELEM_PER_VECTOR ) * (ELEM_PER_VECTOR * ratio) + ( threadIdx.x % ELEM_PER_VECTOR );
	for(unsigned i = 0; i < ratio; ++i )
		sharedMatA[ accessID + i * ELEM_PER_VECTOR ] = ( ( float* )matA )[ accessID + i * ELEM_PER_VECTOR ];
	__syncthreads( );

	for( unsigned i = 0; i < ELEM_PER_VECTOR; ++i )
	{
		sharedVecY[ threadIdx.x ] = 0.0f;
		atomicAdd( &sharedVecY[ threadIdx.x ], sharedMatA[ i * ELEM_PER_VECTOR + eid ] * vecX[ vid * ELEM_PER_VECTOR + eid ] );
		vecY[ vid * ELEM_PER_VECTOR + i ] = sharedVecY[ threadIdx.x ];

		//vecY[ vid * ELEM_PER_VECTOR + i ] = 0.0f;
		//atomicAdd( &vecY[ vid * ELEM_PER_VECTOR + i ], sharedMatA[ i * ELEM_PER_VECTOR + eid ] * vecX[ vid * ELEM_PER_VECTOR + eid ] );
	}
}

// Coded by 20141500
// Wrong calculation
__global__ void gpu_haaam_1D( OUT float *y, IN float( *M )[ ELEM_PER_VECTOR ], IN float *x )
{
	unsigned j = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned n = blockDim.x * gridDim.x;
	//if( j >= n )
	//	return;

	for( unsigned i = 0; i<ELEM_PER_VECTOR; ++i ) {
		float result = 0;
		for( unsigned k = 0; k<ELEM_PER_VECTOR; ++k ) {
			result += M[ i ][ k ] * x[ k * n + j ];
		}
		y[ i * n + j ] = result;
	}
}