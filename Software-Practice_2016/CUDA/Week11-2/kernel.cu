#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <assert.h>


#define N_EQUATIONS (1<<25)

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START() { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif
		
#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL( hipEventDestroy( cuda_timer_start ) );
	CUDA_CALL( hipEventDestroy( cuda_timer_stop ) );
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif


__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if (hipPeekAtLastError() != hipSuccess)
	{
		printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

#define MAX_N_ELEMENTS	(1 << 25)

void generate_random_float_array(float *array, int n) {

	int i;

	for (i = 0; i < n; i++) {
		array[i] = 3.1415926f*((float)rand() / RAND_MAX);
	}
}

// gpu code
void find_roots_CPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n)
{
	int i;
	float a, b, c, d, x0, x1, tmp;
	for (i = 0; i < n; i++)
	{
		a = A[i]; b = B[i]; c = C[i];
		d = sqrtf(b*b - 4.0f*a*c);
		tmp = 1.0f / (2.0f*a);
		X0[i] = x0 = (-b - d) * tmp;
		X1[i] = x1 = (-b + d) * tmp;
		FX0[i] = (a*x0 + b)*x0 + c;
		FX1[i] = (a*x1 + b)*x1 + c;
	}
}

__global__ void find_roots_GPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1)
{
	unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	float a,b,c,d,x0,x1,tmp;

	a = A[tid]; b = B[tid], c = C[tid];
	d = sqrtf(b*b - 4.0f*a*c);
	tmp = 1.0f / (2.0f*a);
	X0[tid] = x0 = (-b - d) * tmp;
	X1[tid] = x1 = (-b + d) * tmp;
	FX0[tid] = (a*x0 + b)*x0 + c;
	FX1[tid] = (a*x1 + b)*x1 + c;
}

void main(void) {

	TIMER_T compute_time = 0;
	TIMER_T device_time = 0;
	
	FILE *fp = fopen( "abc.bin", "rb" );

	int n;
	fread( &n, sizeof( int ), 1, fp );
	float *A = new float[ N_EQUATIONS ];
	float *B = new float[ N_EQUATIONS ];
	float *C = new float[ N_EQUATIONS ];
	float *X0 = new float[ N_EQUATIONS ];
	float *X1 = new float[ N_EQUATIONS ];
	float *FX0 = new float[ N_EQUATIONS ];
	float *FX1 = new float[ N_EQUATIONS ];
	
	float *CPU_X0 = new float[ N_EQUATIONS ];
	float *CPU_X1 = new float[ N_EQUATIONS ];
	float *CPU_FX0 = new float[ N_EQUATIONS ];
	float *CPU_FX1 = new float[ N_EQUATIONS ];
	fread( A, sizeof( float ), N_EQUATIONS, fp );
	fread( B, sizeof( float ), N_EQUATIONS, fp );
	fread( C, sizeof( float ), N_EQUATIONS, fp );
	
	fclose(fp);

	printf( "*** CPU Works...\n" );
	CHECK_TIME_START();
	find_roots_CPU(A,B,C,CPU_X0,CPU_X1,CPU_FX0,CPU_FX1,n);
	CHECK_TIME_END( compute_time );
	printf( " - Finish\n\n" );

	CUDA_CALL(hipSetDevice(0));
	
	float *cudaA,*cudaB,*cudaC,*cudaX0,*cudaX1,*cudaFX0,*cudaFX1;
	CUDA_CALL(hipMalloc(&cudaA, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaB, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaC, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaX0, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaX1, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaFX0, sizeof(float) * N_EQUATIONS));
	CUDA_CALL(hipMalloc(&cudaFX1, sizeof(float) * N_EQUATIONS));

	printf("*** Copying A and B and C from host to device...\n");
	CUDA_CALL(hipMemcpy(cudaA, A, sizeof(float) * N_EQUATIONS, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(cudaB, B, sizeof(float) * N_EQUATIONS, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(cudaC, C, sizeof(float) * N_EQUATIONS, hipMemcpyHostToDevice));
	printf(" - Finish\n\n");

	CHECK_TIME_INIT_GPU();

	size_t n_threads = (1<<10);
	size_t n_blocks = n / n_threads;

	printf("*** kernel call: Get X0 X1 FX0 FX1 <<< %d, %d >>>()...\n", n_blocks, n_threads);
	CHECK_TIME_START_GPU();
	find_roots_GPU<<<n_blocks, n_threads >>>(cudaA,cudaB,cudaC,cudaX0,cudaX1,cudaFX0,cudaFX1);
	cuda_error_check("- ", " FAILED:  X0 X1 FX0 FX()\n\n");
	CHECK_TIME_END_GPU(device_time);
	printf( " - Finish\n\n" );

	printf("*** Time taken = %.6fms(CPU), %.6fms(GPU)\n", compute_time, device_time);

	printf("*** Copying Y from device to host...\n");
	CUDA_CALL(hipMemcpy(X0, cudaX0, sizeof(float) * N_EQUATIONS, hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(X1, cudaX1, sizeof(float) * N_EQUATIONS, hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(FX0, cudaFX0, sizeof(float) * N_EQUATIONS, hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(FX1, cudaFX1, sizeof(float) * N_EQUATIONS, hipMemcpyDeviceToHost));
	CUDA_CALL( hipDeviceSynchronize() );
	/* ���� */
	int cnt = 0;
	for( int i = 0; i < N_EQUATIONS ; ++i )
	{
		if(  fabs(X0[ i ] - CPU_X0[ i ]) > 0.00001 )
			cnt++;
		
	}
	printf( "X0 - %.2f%% numerical errors...\n", cnt/(float)(N_EQUATIONS)*100 );

	 cnt = 0;
	for( int i = 0; i < N_EQUATIONS ; ++i )
	{
		if(fabs(X1[ i ] - CPU_X1[ i ]) > 0.00001 )
			cnt++;
		
	}
	printf( "X1 - %.2f%% numerical errors...\n", cnt/(float)(N_EQUATIONS)*100 );

	cnt = 0;
	for( int i = 0; i < N_EQUATIONS ; ++i )
	{
		if( fabs(FX0[ i ] - CPU_FX0[ i ]) > 0.00001 )
			cnt++;
		
	}
	printf( "FX0 - %.2f%% numerical errors...\n", cnt/(float)(N_EQUATIONS)*100 );
	
	cnt = 0;
	for( int i = 0; i < N_EQUATIONS ; ++i )
	{
		if( fabs(FX1[ i ] - CPU_FX1[ i ]) > 0.00001 )
			cnt++;
		
	}
	printf("FX1 - %.2f%% numerical errors...\n", cnt/(float)(N_EQUATIONS)*100 );
	
	printf(" - Finish\n\n");

	FILE *fp1 = fopen( "X0.bin", "wb" );
	fwrite( X0, sizeof( float ), N_EQUATIONS, fp1);
	fclose(fp1);

	FILE *fp2 = fopen( "X1.bin", "wb" );
	fwrite( X1, sizeof( float ), N_EQUATIONS, fp2);
	fclose(fp2);

	FILE *fp3 = fopen( "FX0.bin", "wb" );
	fwrite( FX0, sizeof( float ), N_EQUATIONS, fp3);
	fclose(fp3);

	FILE *fp4 = fopen( "FX1.bin", "wb" );
	fwrite( FX1, sizeof( float ), N_EQUATIONS, fp4);
	fclose(fp4);
	
	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
	hipFree(cudaX0);
	hipFree(cudaX1);
	hipFree(cudaFX0);
	hipFree(cudaFX1);

	CHECK_TIME_DEST_GPU();
	
	CUDA_CALL(hipDeviceReset());

	delete[] A;
	delete[] B;
	delete[] C;
	delete[] X0;
	delete[] X1;
	delete[] FX0;
	delete[] FX1;

}