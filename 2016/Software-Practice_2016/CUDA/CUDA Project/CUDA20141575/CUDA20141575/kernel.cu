#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <assert.h>


const int ELEM_PER_VECTOR = 32;
#define ELEM_PER_POINT (1 << 5)
#define N_ELEMS (1<<22)
#define N_POINTS (N_ELEMS / ELEM_PER_POINT)

typedef struct
{
	float *elem[ELEM_PER_POINT];
}POINTS_SOA;

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START() { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

inline float absIEEE754( float f)
{
	return ( float& )( ( int& )f &= 0x7fffffff );
}

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif


__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if (hipPeekAtLastError() != hipSuccess)
	{
		printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		exit(1);
	}
}

#define MAX_N_ELEMENTS	(1 << 25)

/* CPU */
void matrix_multiplication_cpu(float *vecY, float (*matA), float *vecX, int n)
{
	for (int i = 0; i<n; ++i)
	{
		for (int j = 0; j<ELEM_PER_VECTOR; ++j)
		{
			vecY[i*ELEM_PER_VECTOR + j] = 0;
			for (int k = 0; k<ELEM_PER_VECTOR; ++k)
				vecY[i*ELEM_PER_VECTOR + j] += matA[j*ELEM_PER_VECTOR + k] * vecX[i*ELEM_PER_VECTOR + k];
		}
	}
}
/* GPU1 */
__global__ void matrix_multiplication_gpu_global_memory(float* vecY, float(*matA), float* vecX)
{
	unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	float result;
	for (unsigned i = 0; i < ELEM_PER_VECTOR; ++i)
	{
		result = 0.0f;
		for (unsigned j = 0; j < ELEM_PER_VECTOR; ++j)
		{
			result += matA[i*ELEM_PER_VECTOR + j] * vecX[tid * ELEM_PER_VECTOR + j];
		}
		vecY[tid * ELEM_PER_VECTOR + i] = result;
	}
}
/* GPU2 */
__global__ void matrix_multiplication_gpu_shared_memory(float* vecY, float* matA, float* vecX)
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = ( tid % ELEM_PER_VECTOR ) + ( tid / 1024 * ELEM_PER_VECTOR ); // blockDim.x
	unsigned eid = ( tid % 1024 / ELEM_PER_VECTOR );

	__shared__ float sharedVecX[ 1024 ];
	__shared__ float sharedMatA[ 1024 ];

	unsigned ratio = 1024 / blockDim.x; // num of elements in 32x32 Matrix
	int accessID = ( threadIdx.x / ELEM_PER_VECTOR ) * ( ELEM_PER_VECTOR * ratio ) + ( threadIdx.x % ELEM_PER_VECTOR );
	for( unsigned i = 0; i < ratio; ++i )
		sharedMatA[ accessID + i * ELEM_PER_VECTOR ] = ( ( float* )matA )[ accessID + i * ELEM_PER_VECTOR ];

	sharedVecX[ threadIdx.x ] = vecX[ tid ];
	unsigned svid = threadIdx.x % ELEM_PER_VECTOR * ELEM_PER_VECTOR;
	__syncthreads( );

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += sharedMatA[ eid * ELEM_PER_VECTOR + j ] * sharedVecX[ svid + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}
/* GPU3*/
__global__ void matrix_multiplication_gpu_SOA_shared_memory(POINTS_SOA vecY, float* matA, POINTS_SOA vecX)
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;

	__shared__ float sharedVecX[ 1024 ];
	__shared__ float sharedMatA[ 1024 ];                

	unsigned ratio = 1024 / blockDim.x; // num of elements in 32x32 Matrix
	int accessID = ( threadIdx.x / ELEM_PER_VECTOR ) * ( ELEM_PER_VECTOR * ratio ) + ( threadIdx.x % ELEM_PER_VECTOR );
	for( unsigned i = 0; i < ratio; ++i )
		sharedMatA[ accessID + i * ELEM_PER_VECTOR ] = ( ( float* )matA )[ accessID + i * ELEM_PER_VECTOR ];

	sharedVecX[ threadIdx.x ] = vecX.elem[eid][ vid ];
	unsigned svid = threadIdx.x / ELEM_PER_VECTOR * ELEM_PER_VECTOR; 
	__syncthreads();

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += sharedMatA[ eid * ELEM_PER_VECTOR + j ] * sharedVecX[ svid + j ];
	}
	vecY.elem[eid][vid] = result;
}

/* GPU4*/
__global__ void matrix_multiplication_gpu_optimized(float* vecY, float* matA, float* vecX)
{
	unsigned tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned vid = tid / ELEM_PER_VECTOR;
	unsigned eid = tid % ELEM_PER_VECTOR;
	//unsigned vid = ( tid % ELEM_PER_VECTOR ) + ( tid / 1024 * ELEM_PER_VECTOR ); // blockDim.x
	//unsigned eid = ( tid % 1024 / ELEM_PER_VECTOR );

	__shared__ float sharedVecX[ 1024 ];
	__shared__ float sharedMatA[ 1024 ];

	unsigned ratio = 1024 / blockDim.x; // num of elements in 32x32 Matrix
	int accessID = ( threadIdx.x / ELEM_PER_VECTOR ) * ( ELEM_PER_VECTOR * ratio ) + ( threadIdx.x % ELEM_PER_VECTOR );
	int accessID_transpose = ( threadIdx.x % ELEM_PER_VECTOR ) * ( ELEM_PER_VECTOR * ratio ) + ( threadIdx.x / ELEM_PER_VECTOR );

	for( unsigned i = 0; i < ratio; ++i )
		sharedMatA[ accessID_transpose + i * ELEM_PER_VECTOR ] = ( ( float* )matA )[ accessID + i * ELEM_PER_VECTOR ];

	sharedVecX[ threadIdx.x ] = vecX[ tid ];
	unsigned svid = threadIdx.x / ELEM_PER_VECTOR * ELEM_PER_VECTOR;
	__syncthreads( );

	float result = 0.0f;
	for( unsigned j = 0; j < ELEM_PER_VECTOR; ++j )
	{
		result += sharedMatA[ j * ELEM_PER_VECTOR + eid ] * sharedVecX[ svid + j ];
	}
	vecY[ vid * ELEM_PER_VECTOR + eid ] = result;
}

void main(void) {
	int n_elements;
	TIMER_T compute_time = 0;
	TIMER_T device_time_1 = 0;
	TIMER_T device_time_2 = 0;
	TIMER_T device_time_3 = 0;
	TIMER_T device_time_4 = 0;
	FILE *fp = fopen("gen.bin", "rb");

	/* initialize array & variable*/
	float *mat  = new float[ELEM_PER_VECTOR * ELEM_PER_VECTOR];
	int n;
	fread(&n, sizeof(float), 1, fp);
	float *vecX = new float[ELEM_PER_VECTOR * n];
	fread(vecX, sizeof(float), n * ELEM_PER_VECTOR, fp);
	fread(mat, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);

	fclose(fp);
	
	float *vecY_cpu  = new float[n * ELEM_PER_VECTOR];
	float *vecY_1 = new float[n * ELEM_PER_VECTOR]; //y1
	float *vecY_2 = new float[n * ELEM_PER_VECTOR]; //y2
	float *vecY_3 = new float[n * ELEM_PER_VECTOR]; //y3
	float *vecY_4 = new float[n * ELEM_PER_VECTOR]; //y4

	/* CPU exec */
	CHECK_TIME_START();
	matrix_multiplication_cpu(vecY_cpu,mat,vecX,n);
	CHECK_TIME_END(compute_time);
	/* CPU output*/
	printf("Elapsed Time by CPU is %.6f(s)\n",compute_time/1000);


	/* initialize Cuda variable */
	CUDA_CALL(hipSetDevice(0));

	float *cudaY_1, *cudaY_2, *cudaY_4;
	float *cudaX, *cudaMat;
	CUDA_CALL(hipMalloc(&cudaY_1, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaY_2, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaY_4, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaX, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaMat, sizeof(float) * ELEM_PER_VECTOR * ELEM_PER_VECTOR));

	//*** Copying A and B from host to device... copy to Cuda varible 
	CUDA_CALL(hipMemcpy(cudaX, vecX, sizeof(float) * n * ELEM_PER_VECTOR, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(cudaMat, mat, sizeof(float) * ELEM_PER_VECTOR * ELEM_PER_VECTOR, hipMemcpyHostToDevice));
	CHECK_TIME_INIT_GPU();


	size_t n_threads = (1 << 10);
	size_t n_blocks = n / n_threads;

	size_t _1024Threads = ( 1 << 10 );
	size_t _1024Blocks_perElement = ( n * ELEM_PER_VECTOR ) / _1024Threads;
	size_t _1024Blocks_perVector = n / _1024Threads;

	/* GPU1 exec*/
	CHECK_TIME_START_GPU();
	matrix_multiplication_gpu_global_memory <<<_1024Blocks_perVector, _1024Threads >>>(cudaY_1, cudaMat, cudaX);
	cuda_error_check("- ", " FAILED: matrix_multiplication_gpu_global_memory()\n\n");
	CHECK_TIME_END_GPU(device_time_1);

	/* GPU2 exec*/
	CHECK_TIME_START_GPU();
	matrix_multiplication_gpu_shared_memory <<<_1024Blocks_perElement, _1024Threads >>>(cudaY_2, cudaMat, cudaX);
	cuda_error_check("- ", " FAILED: matrix_multiplication_gpu_shared_memory()\n\n");
	CHECK_TIME_END_GPU(device_time_2);
	
	/* GPU3 exec*/
	POINTS_SOA CudaX3,CudaY3,SoaX,SoaY;
	for(int i = 0 ; i < ELEM_PER_VECTOR ; ++i)
	{
		SoaX.elem[i] = (float*)malloc(sizeof(float) * n);
		SoaY.elem[i] = (float*)malloc(sizeof(float) * n);
		CUDA_CALL(hipMalloc(&CudaX3.elem[i],sizeof(float) * n));
		CUDA_CALL(hipMalloc(&CudaY3.elem[i],sizeof(float) * n));
	}

	for(int i = 0 ; i < n ; ++i)
	{
		for(int j = 0 ; j < ELEM_PER_VECTOR ; ++j)
		{
			SoaX.elem[j][i] = vecX[i * ELEM_PER_VECTOR + j];
		}
	}
	for(int i = 0 ; i < ELEM_PER_VECTOR ; ++i)
		CUDA_CALL(hipMemcpy(CudaX3.elem[i], SoaX.elem[i], sizeof(float) * n, hipMemcpyHostToDevice));

	CHECK_TIME_START_GPU();
	matrix_multiplication_gpu_SOA_shared_memory <<<_1024Blocks_perElement, _1024Threads >>>(CudaY3,cudaMat,CudaX3);
	cuda_error_check("- ", " FAILED: matrix_multiplication_gpu_SOA_shared_memory()\n\n");
	CHECK_TIME_END_GPU(device_time_3);
	
	/* GPU4 exec*/
	CHECK_TIME_START_GPU();
	matrix_multiplication_gpu_optimized <<<_1024Blocks_perElement, _1024Threads >>>(cudaY_4,cudaMat,cudaX);
	cuda_error_check("- ", " FAILED: matrix_multiplication_gpu_optimized()\n\n");
	CHECK_TIME_END_GPU(device_time_4);

	/* *** Copying Y from device to host...*/
	CUDA_CALL(hipMemcpy(vecY_1, cudaY_1, sizeof(float)*n * ELEM_PER_VECTOR, hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(vecY_2, cudaY_2, sizeof(float)*n * ELEM_PER_VECTOR, hipMemcpyDeviceToHost));
	for(int i = 0 ; i < ELEM_PER_VECTOR ; ++i)
		CUDA_CALL(hipMemcpy(SoaY.elem[i], CudaY3.elem[i], sizeof(float) * n,hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(vecY_4, cudaY_4, sizeof(float)*n * ELEM_PER_VECTOR, hipMemcpyDeviceToHost));
	CUDA_CALL(hipDeviceSynchronize());

	/* GPU1 output*/
	float EPSILON = 0.0001f;
	int cnt = 0;
	for (int i = 0; i < n * ELEM_PER_VECTOR; ++i)
	{
		if ( fabs(vecY_cpu[i] - vecY_1[i]) > EPSILON )
		{
			cnt++;
		}
	}

	printf("Elapsed Time by GPU1 is %.6f(s)", device_time_1/1000); 
	printf(" Error rate is %.2f%%\n", cnt / (float)(n * ELEM_PER_VECTOR) * 100);

	/* GPU2 output*/
	cnt = 0;
	for (int i = 0; i < n * ELEM_PER_VECTOR; ++i)
	{
		if( fabs(vecY_cpu[i] - vecY_2[i]) > EPSILON)		
		{
			cnt++;
		}
	}
	printf("Elapsed Time by GPU2 is %.6f(s)",device_time_2/1000);
	printf(" Error rate is %.2f%%\n", cnt / (float)(n * ELEM_PER_VECTOR) * 100);

	/* GPU3 output*/
	cnt = 0;
	for (int i = 0; i < n ; ++i)
	{
		for(int j = 0 ; j < ELEM_PER_VECTOR; ++j)
		{
			if (absIEEE754(SoaY.elem[j][i] - vecY_cpu[i*ELEM_PER_VECTOR + j]) > EPSILON)
			{
				cnt++;
			}
		}
	}
	printf("Elapsed Time by GPU3 is %.6f(s)",device_time_3/1000);
	printf(" Error rate is %.2f%%\n", cnt / (float)(n * ELEM_PER_VECTOR) * 100);

	/* GPU4 output*/
	cnt = 0;
	for (int i = 0; i < n * ELEM_PER_VECTOR; ++i)
	{
		if (fabs(vecY_cpu[i] - vecY_4[i]) > EPSILON)
		{
			cnt++;
		}
	}

	printf("Elapsed Time by GPU4 is %.6f(s)",device_time_4/1000);
	printf(" Error rate is %.2f%%\n", cnt / (float)(n * ELEM_PER_VECTOR) * 100);

	/* free */
	hipFree(cudaY_1);
	hipFree(cudaY_2);
	hipFree(cudaY_4);
	hipFree(cudaX);
	hipFree(cudaMat);

	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipDeviceReset());

	delete[] vecX;
	delete[] vecY_cpu;
	delete[] vecY_1;
	delete[] vecY_2;
	delete[] vecY_3;
	delete[] vecY_4;
	delete[] mat;
}