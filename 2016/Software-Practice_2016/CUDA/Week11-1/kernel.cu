#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <assert.h>

const int ELEM_PER_VECTOR = 32;

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START() { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL( hipEventDestroy( cuda_timer_start ) );
	CUDA_CALL( hipEventDestroy( cuda_timer_stop ) );
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif


__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if (hipPeekAtLastError() != hipSuccess)
	{
		printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

#define MAX_N_ELEMENTS	(1 << 25)

void generate_random_float_array(float *array, int n) {

	int i;

	for (i = 0; i < n; i++) {
		array[i] = 3.1415926f*((float)rand() / RAND_MAX);
	}
}

// gpu code
__global__ void combine_two_arrays_GPU( float* x, float* y, float* z )
{
	unsigned int tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	z[ tid ] = 1.0f / (sinf( x[ tid ] )*cosf( y[ tid ] ) + cosf( x[ tid ] )*sinf( y[ tid ] ) );
}

void combine_two_arrays_CPU( float *x, float *y, float *z, int n )
{
	int i;

	for( i = 0; i < n; i++ )
	{
		//z[ i + 0 ] = 1.0f / i;
		z[ i + 0 ] = 1.0f / ( sinf( x[ i + 0 ] )*cosf( y[ i + 0 ] ) + cosf( x[ i + 0 ] )*sinf( y[ i + 0 ] ) );
	}
}

__global__ void combine_two_arrays_GPU_matrix(float *y, float *mat, float *x)
{
	unsigned int tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	for(int j=0;j<ELEM_PER_VECTOR;++j)
	{
		y[tid*ELEM_PER_VECTOR + j] = 0;
		for(int k=0;k<ELEM_PER_VECTOR;++k)
			y[tid*ELEM_PER_VECTOR + j] += mat[j*ELEM_PER_VECTOR + k] * x[tid*ELEM_PER_VECTOR + k];
	}
}

void combine_two_arrays_CPU_matrix(float *y,float *mat, float *x,int n)
{
	for(int i=0;i<n;++i)
	{
		for(int j=0;j<ELEM_PER_VECTOR;++j)
		{
			y[i*ELEM_PER_VECTOR + j] = 0;
			for(int k=0;k<ELEM_PER_VECTOR;++k)
				y[i*ELEM_PER_VECTOR + j] += mat[j*ELEM_PER_VECTOR + k] * x[i*ELEM_PER_VECTOR + k];
		}
	}
}

void main(void) {
	int n_elements;

	TIMER_T compute_time = 0;
	TIMER_T device_time = 0;

	FILE *fp = fopen( "gen.bin", "rb" );

	float *mat = new float[ ELEM_PER_VECTOR * ELEM_PER_VECTOR ];
	int n;
	fread( &n, sizeof( float ), 1, fp );
	float *x = new float[ ELEM_PER_VECTOR * n ];
	fread( x, sizeof( float ), n * ELEM_PER_VECTOR, fp );
	fread( mat, sizeof( float ), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp );

	fclose(fp);

	float *y1 = new float[ n * ELEM_PER_VECTOR ]; //y1
	float *y2 = new float[ n * ELEM_PER_VECTOR ]; //y2

	printf( "*** CPU Works...\n" );
	CHECK_TIME_START();
	combine_two_arrays_CPU_matrix(y1, mat, x, n);
	CHECK_TIME_END( compute_time );
	printf( " - Finish\n\n" );

	CUDA_CALL(hipSetDevice(0));

	float *cudaY, *cudaX, *cudaMat;
	CUDA_CALL(hipMalloc(&cudaY, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaX, sizeof(float) * n * ELEM_PER_VECTOR));
	CUDA_CALL(hipMalloc(&cudaMat, sizeof(float) * ELEM_PER_VECTOR * ELEM_PER_VECTOR));

	printf("*** Copying A and B from host to device...\n");
	CUDA_CALL(hipMemcpy(cudaX, x, sizeof(float) * n * ELEM_PER_VECTOR, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(cudaMat, mat, sizeof(float) * ELEM_PER_VECTOR * ELEM_PER_VECTOR, hipMemcpyHostToDevice));
	printf(" - Finish\n\n");

	CHECK_TIME_INIT_GPU();

	size_t n_threads = (1<<10);
	size_t n_blocks = n / n_threads;

	printf("*** kernel call: combine_two_arrays<<< %d, %d >>>()...\n", n_blocks, n_threads);
	CHECK_TIME_START_GPU();
	combine_two_arrays_GPU_matrix<<<n_blocks, n_threads >>>(cudaY, cudaMat, cudaX);
	cuda_error_check("- ", " FAILED: combine_two_arrays()\n\n");
	CHECK_TIME_END_GPU(device_time);
	printf( " - Finish\n\n" );

	printf("*** Time taken = %.6fms(CPU), %.6fms(GPU)\n", compute_time, device_time);

	printf("*** Copying Y from device to host...\n");
	CUDA_CALL(hipMemcpy(y2, cudaY, sizeof(float)*n * ELEM_PER_VECTOR, hipMemcpyDeviceToHost));
	CUDA_CALL( hipDeviceSynchronize() );
	
	int cnt = 0;
	for( int i = 0; i < n * ELEM_PER_VECTOR; ++i )
	{
		if( y1[ i ] != y2[ i ] )
		{
			//fprintf( fp, "[%d] %f != %f\n\n", i, C1[i], C2[i] );
			//printf("[%d] %f != %f\n\n", i, C1[ i ], C2[ i ] );
			cnt++;
		}
	}
	//fclose( fp );

	printf( " - %.2f%% numerical errors...\n", cnt/(float)(n * ELEM_PER_VECTOR)*100 );
	printf(" - Finish\n\n");
	
	hipFree(cudaY);
	hipFree(cudaX);
	hipFree(cudaMat);

	CHECK_TIME_DEST_GPU();
	
	CUDA_CALL(hipDeviceReset());

	delete[] x;
	delete[] y1;
	delete[] y2;
	delete[] mat;
}